#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "ppm_file_reader.h"
#include "kernels.h"

//base of this code is taken from Roland Konlechner HPC 2021 assignment 4: CUDA convolution kernel
//make one convolution engine which accepts arbitrary 9x9 kernels
__global__ void Convolution(PPMPixel* img_in, PPMPixel* img_out, float* kernel, int size_x, int size_y)
{
    int globalIdx = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    int nr_blocks = blockDim.x * blockDim.y * blockDim.z;

    globalIdx += nr_blocks*(blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x);

    //divide rows of image amongst blocks, and colums amongst threads within blocks
    float heightPerBlock = (float)size_y / (float)gridDim.x;
    float widthPerThread = (float)size_x / (float)blockDim.x;

    int blockStart = round(blockIdx.x * heightPerBlock);
    int blockStop  = round((blockIdx.x+1) * heightPerBlock - 1);
    int threadStart = round(threadIdx.x * widthPerThread);
    int threadStop = round((threadIdx.x+1) * widthPerThread - 1);

    //allocate local kernel buffer and copy part of image. For the edge cases, fill the buffer with black
    PPMPixel conv_buffer[81];
    float sum_red;
    float sum_green;
    float sum_blue;

    int k_offset = 4;
    int k_size = 9;

    for(int row=blockStart; row<=blockStop; row++)
    {
        for(int col=threadStart; col<=threadStop; col++)
        {
            for(int kern_row=-k_offset; kern_row<=k_offset; kern_row++)
            {
                for(int kern_col=-k_offset; kern_col<=k_offset; kern_col++)
                {
                    if(row+kern_row < 0 || row+kern_row > size_y || col+kern_col < 0 || col+kern_col > size_x)
                    {                        
                        conv_buffer[(kern_row+k_offset)*k_size+(kern_col+k_offset)].red   = 0;
                        conv_buffer[(kern_row+k_offset)*k_size+(kern_col+k_offset)].green = 0;
                        conv_buffer[(kern_row+k_offset)*k_size+(kern_col+k_offset)].blue  = 0;
                    }else{
                        conv_buffer[(kern_row+k_offset)*k_size+(kern_col+k_offset)].red   = img_in[(row+kern_row)*size_x+(col+kern_col)].red;
                        conv_buffer[(kern_row+k_offset)*k_size+(kern_col+k_offset)].green = img_in[(row+kern_row)*size_x+(col+kern_col)].green;
                        conv_buffer[(kern_row+k_offset)*k_size+(kern_col+k_offset)].blue  = img_in[(row+kern_row)*size_x+(col+kern_col)].blue;
                    }
                }
            }
            sum_red     = 0.0;
            sum_green   = 0.0;
            sum_blue    = 0.0;
            
            //do the convolution on the local buffer and write result to output
            for(int i=0; i<81; i++)
            {
                sum_red    += (float)conv_buffer[i].red   * kernel[i];
                sum_green  += (float)conv_buffer[i].green * kernel[i];
                sum_blue   += (float)conv_buffer[i].blue  * kernel[i];
            }
            img_out[row*size_x+col].red     = (char)sum_red;
            img_out[row*size_x+col].green   = (char)sum_green;
            img_out[row*size_x+col].blue    = (char)sum_blue;
        }
    }
}


int main(){
    PPMImage* h_img[3];
    h_img[0] = readPPM("../data/8k_mountains.ppm");
    h_img[1] = readPPM("../data/16k_forest.ppm");
    h_img[2] = readPPM("../data/32k_death_valley.ppm");

    char* outputNames[3] = {"../data/8k_mountains_blurry_cuda.ppm", "../data/16k_forest_blurry_cuda.ppm", "../data/32k_death_valley_blurry_cuda.ppm"};

    // from https://stackoverflow.com/questions/28112485/how-to-select-a-gpu-with-cuda/28113186

    hipSetDevice(1);

    hipEvent_t start_copy, stop_copy, start_no_copy, stop_no_copy;

    hipEventCreate(&start_copy);
    hipEventCreate(&start_no_copy);
    hipEventCreate(&stop_copy);
    hipEventCreate(&stop_no_copy);

    for(int i=0; i<3; i++)
    {
        printf("Test run %d\n", i);

        long int img_size = h_img[i]->x * h_img[i]->y * sizeof(PPMPixel);
        int kernel_size = 9*9*sizeof(float);

        PPMPixel* d_img_data;
        PPMPixel* d_res_data;
        PPMPixel* h_res_data;

        float* h_kernel_box_blur = kernel_box_blur_9x9();
        float* d_kernel_box_blur;

        h_res_data = (PPMPixel*)malloc(img_size);
        hipMalloc(&d_img_data, img_size);
        hipMalloc(&d_res_data, img_size);
        hipMalloc(&d_kernel_box_blur, kernel_size);

        hipEventRecord(start_copy);

        hipMemcpy(d_img_data, h_img[i]->data, img_size, hipMemcpyHostToDevice);
        hipMemcpy(d_kernel_box_blur, h_kernel_box_blur, kernel_size, hipMemcpyHostToDevice);

        //data on host is not needed anymore, let host image reference the result data
        free(h_img[i]->data);
        h_img[i]->data = h_res_data;

        hipEventRecord(start_no_copy);

        Convolution<<<32, 1024>>>(d_img_data, d_res_data, d_kernel_box_blur, h_img[i]->x, h_img[i]->y);
        
        hipEventRecord(stop_no_copy);
        
        hipMemcpy(h_res_data, d_res_data, img_size, hipMemcpyDeviceToHost);
        
        hipEventRecord(stop_copy);
        
        hipEventSynchronize(stop_no_copy);
        hipEventSynchronize(stop_copy);
        hipDeviceSynchronize();

        float milliseconds_copy = 0;
        float milliseconds_no_copy = 0;

        hipEventElapsedTime(&milliseconds_copy, start_copy, stop_copy);
        hipEventElapsedTime(&milliseconds_no_copy, start_no_copy, stop_no_copy);

        writePPM(outputNames[i], h_img[i]);

        printf("\tTime elapsed (no copy) [ms]:\t%f\n", milliseconds_no_copy);
        printf("\tTime elapsed (copy) [ms]:\t%f\n", milliseconds_copy);

        free(h_kernel_box_blur);
        free(h_res_data);
        hipFree(d_img_data);
        hipFree(d_res_data);
        hipFree(h_kernel_box_blur);
    }

    free(h_img[0]);
    free(h_img[1]);
    free(h_img[2]);

